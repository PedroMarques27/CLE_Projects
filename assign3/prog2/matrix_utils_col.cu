/**
 *  \file matrix_utils.c
 *
 *  \brief Problem name: Matrix Determinant Calculation With CUDA.

 *  Utility functions to calculate the determinant of a matrix
 *
 *  \author Mário Silva, Pedro Marques - June 2022
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <math.h>


/**
 *  \brief 
 *  Calculates the determinant of a given matrix
 *  \param matrix the matrix to be processed
 *  \param argv order of the matrix
 *
 *  \return the determinant of the matrix
 */
double getDeterminant(int order, double *matrix){
    int i,j,k;
    int swaps = 0;
    for(i=0;i<order-1;i++){
        //Partial Pivoting
        for(k=i+1;k<order;k++){
            //If diagonal element(absolute vallue) is smaller than any of the terms below it
            if(fabs(*((matrix+i*order) + i))<fabs(*((matrix+k*order) + i))){
                //Swap the rows
                swaps++;
                for(j=0;j<order;j++){                
                    double temp;
                    temp=*((matrix+j*order) + i);
                    *((matrix+j*order) + i)=*((matrix+j*order) + k);
                    *((matrix+j*order) + k)=temp;
                }
            }
        }
        //Begin Gauss Elimination
        for(k=i+1;k<order;k++){
            double  term=*((matrix+i*order) + k)/ *((matrix+i*order) + i);
            for(j=0;j<order;j++){
                *((matrix+j*order) + k)=*((matrix+j*order) + k)-term*(*((matrix+j*order) + i));
            }
        }
    }
    
    double det = 1;
    for(int i=0; i<order; i++){
        det *= (*((matrix+i*order) + i));
    }
	return pow(-1,swaps)*det;;    
}

/**
 *  \brief 
 *  For a given column, calculates the pivot and continuously updates the determinant's value
 *  \param matricesDevice pointer to the matrix in the device
 *  \param orderDevice pointer to the order of the matrix in the device
 *  \param determinants pointer to the array of determinants in the device
 *  \param currentCol current column matrix index whose pivot is being determined
 * 
 */
__global__ void calcPivots(double *matricesDevice, int *orderDevice, double *determinants, int *currentCol)
{
  int iteration = *currentCol;
  if (threadIdx.x == iteration)
  {
    int order = *orderDevice;
    bool switchedCols = false;
    double *matrix = matricesDevice + blockIdx.x * order * order;

    double pivot =  *(matrix + iteration * order + iteration);
    // finding the pivot
    if (pivot == 0.0)
    {
      for (int k = iteration + 1; k < order; k++)
      {
        if ((matrix + iteration*order + k) != 0)
        {
          // Swap the two cols
          for (int j = 0; j < order; j++)
          {
            double temp = *(matrix + iteration * order + iteration);
            *(matrix + j * order + iteration) = *(matrix + j * order + k);
            *(matrix + j * order + k) = temp;
          }
          switchedCols = true;
          break;
        }
      }
    }

    pivot = *(matrix + iteration * order + iteration);

    // calculate the determinants
    if (iteration == 0)
      determinants[blockIdx.x] = pivot;
    else
      determinants[blockIdx.x] *= pivot;

    if (switchedCols)
      determinants[blockIdx.x] *= -1;
  }
}

/**
 *  \brief 
 *  For a given column, subtracts the pivot calculated with calcPivots, executing Gauss Elimination
 *  \param matricesDevice pointer to the matrix in the device
 *  \param orderDevice pointer to the order of the matrix in the device
 *  \param determinants pointer to the array of determinants in the device
 *  \param currentCol current column matrix index to be subtracted the pivot
 * 
 */
__global__ void subtractPivots(double *matricesDevice, int *orderDevice, double *determinants, int *currentCol)
{
  int iteration = *currentCol;
  if (threadIdx.x > iteration) {
    int order = *orderDevice;
    double *matrix = matricesDevice + blockIdx.x * order * order;
    double *col = matrix + threadIdx.x;
    double *pivotCol = matrix + iteration;
    double pivot = *(pivotCol + iteration*order);

    double scale = col[iteration*order] / pivot;
    // Begin Gauss Elimination
    for(int k=iteration+1; k<order; k++)
    {
      col[k*order] -= scale * pivotCol[k*order];
    }
  }
}